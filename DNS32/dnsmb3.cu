/*
CUDA DNS Matrix Multiply: Multiblock Experiment 3
4 * WIDTH blocks each 0.25 * WIDTH-by-WIDTH
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <cstring>
#include <math.h>
#include <math.h>       /* pow */
//#include <cutil.h>

#define TILE_WIDTH 8

__global__ void MatMulKernel(float* d_M, float* d_N, float* d_P, int Width) {

	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH];
	__shared__ float partialSum[TILE_WIDTH][TILE_WIDTH];

	int tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x / 4, bm = blockIdx.x % 4;

	Mds[tx][ty] = d_M[(tx + 2 * bm) * TILE_WIDTH + ty];
	if (tx == 0) Nds[ty] = d_N[ty * TILE_WIDTH + bx];
	__syncthreads();

	partialSum[tx][ty] = Mds[tx][ty] * Nds[ty];
	__syncthreads();

	if (ty < 4) {

		partialSum[tx][ty] += partialSum[tx][ty + 4];
		if (ty < 2) partialSum[tx][ty] += partialSum[tx][ty + 2];
		if (ty == 0) d_P[(tx + 2 * bm) * TILE_WIDTH + bx] = partialSum[tx][ty] + partialSum[tx][ty + 1];
	}
}

void MatrixMultiplication(float* M, float* N, float* P, int Width) {

	int size = Width * Width * sizeof(float);
	float *Md, *Nd, *Pd;

	// Transfer M and N to device memory
	hipMalloc((void**) &Md, size);
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
	hipMalloc((void**) &Nd, size);
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

	// Allocate P on the device
	hipMalloc((void**) &Pd, size);

	int blockfactor = pow(8, ((Width / 8) - 1));
	dim3 dimGrid(TILE_WIDTH * 4 * blockfactor, 1, 1); //#blocks
	dim3 dimBlock(TILE_WIDTH / 4, TILE_WIDTH); //#threads

	// Launch the device computation threads
	MatMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, Width);

	// Transfer P from device to host
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
	// Free device matrices
	hipFree(Md); hipFree(Nd); hipFree(Pd);
}

int main(int argc, char* argv[]) {

	int Width = atoi(argv[1]);
	//string dimstr(argv[1]);
	hipEvent_t start, stop;
	float elapsedTime;

	unsigned int size_M = Width * Width;
	unsigned int mem_size_M = size_M * sizeof(float);
	float* hostM = (float*) malloc(mem_size_M);
	unsigned int size_N = Width * Width;
	unsigned int mem_size_N = size_N * sizeof(float);
	float* hostN = (float*) malloc(mem_size_N);
	unsigned int size_P = Width * Width;
	unsigned int mem_size_P = size_P * sizeof(float);
	float* hostP = (float*) malloc(mem_size_P);
	unsigned int size_ref = Width * Width;
	unsigned int mem_size_ref = size_ref * sizeof(float);
	float* ref = (float*) malloc(mem_size_ref);

	const int filenamelength = 14;

	// file io
	FILE *mat1, *mat2, *ans;
	char mat1filename[filenamelength];
	strcpy(mat1filename, argv[1]);
	strcat(mat1filename, "matrix1.txt");
	//printf(mat1filename);
	mat1 = fopen(mat1filename, "r");
	for (int i = 0; i < Width; i++){
		for (int j = 0; j < Width; j++){
			fscanf(mat1, "%f", &hostM[i * Width + j]);
			printf("%f ", hostM[i * Width + j]);
		}	printf("\n");}
	fclose(mat1);

	char mat2filename[filenamelength];
	strcpy(mat2filename, argv[1]);
	strcat(mat2filename, "matrix2.txt");
	mat2 = fopen(mat2filename, "r");
	for (int i = 0; i < Width; i++)
		for (int j = 0; j < Width; j++)
			fscanf(mat2, "%f", &hostN[i * Width + j]);
	fclose(mat2);

	char productmatfilename[filenamelength];
	strcpy(productmatfilename, argv[1]);
	strcat(productmatfilename, "product.txt");
	ans = fopen(productmatfilename, "r");
	for (int i = 0; i < Width; i++)
		for (int j = 0; j < Width; j++)
			fscanf(ans, "%f", &ref[i * Width + j]);
	fclose(ans);

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	MatrixMultiplication(hostM, hostN, hostP, Width);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed time: %3.3f us\n", elapsedTime * 1000);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	for (int i = 0; i < Width; i++)
		for (int j = 0; j < Width; j++)
			if (abs(ref[i * Width + j] - hostP[i * Width + j]) > 0.05)
				printf("Error, coord[%i][%i]: ref = %f p = %f\n", i, j, ref[i*Width+j], hostP[i*Width + j]);

	// clean up memory
	free(hostM); free(hostN); free(hostP); free(ref);
	return 0;
}
